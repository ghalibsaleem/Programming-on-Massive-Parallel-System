#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>



#define BOX_SIZE	23000 /* size of the data box on one dimension            */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
	double x_pos;
	double y_pos;
	double z_pos;
} atom;

typedef struct hist_entry{
	//float min;
	//float max;
	unsigned long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;


bucket * histogram;		/* list of all buckets in the histogram   */
long long	PDH_acnt;	/* total number of data points            */
int num_buckets;		/* total number of buckets in the histogram */
double   PDH_res;		/* value of w                             */
atom * atom_list;		/* list of all data points                */

/* These are for an old way of tracking time */
struct timezone Idunno;	
struct timeval startTime, endTime;


/* 
	distance of two points in the atom_list 
*/
double p2p_distance(int ind1, int ind2) {
	//TODO
	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}

__device__ double p2p_distance(atom * atomList, int ind1, int ind2) {
	
	double x1 = atomList[ind1].x_pos;
	double x2 = atomList[ind2].x_pos;
	double y1 = atomList[ind1].y_pos;
	double y2 = atomList[ind2].y_pos;
	double z1 = atomList[ind1].z_pos;
	double z2 = atomList[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


/* 
	brute-force SDH solution in a single CPU thread 
*/
int PDH_baseline() {
	//TODO
	int i, j, h_pos;
	double dist;
	
	for(i = 0; i < PDH_acnt; i++) {
		for(j = i+1; j < PDH_acnt; j++) {
			dist = p2p_distance(i,j);
			h_pos = (int) (dist / PDH_res);
			histogram[h_pos].d_cnt++;
		} 
	}
	return 0;
}



__global__ void PDH_baseline(bucket *histogram, atom * atomList, double width, int size) {
	int i, j, h_pos;
	double dist;

	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = i + 1;

	for(int l_index = j; l_index < size; l_index++){
		dist = p2p_distance(atomList, i, l_index);
		h_pos = (int) (dist / width);
		atomicAdd(&histogram[h_pos].d_cnt, 1);
	}
	
	__syncthreads();
	/*
	for(i = 0; i < PDH_acnt; i++) {
		for(j = i+1; j < PDH_acnt; j++) {
			dist = p2p_distance(i,j);
			h_pos = (int) (dist / PDH_res);
			histogram[h_pos].d_cnt++;
		} 
	}
	return 0;
	*/
}


__global__ void PDH2D_basline_algorithm3(bucket *histogram, atom *atom_list, long long width, double size, int num_blocks, int num_buckets) {
	
	int block, thread, blockdim, blockid;
	atom left, right;
	
	extern __shared__ bucket sharedHistogram[];
	__shared__ atom sharedAtomList[256];

	blockid = blockIdx.x;	
	blockdim = blockDim.x;
	thread = threadIdx.x;
	block = blockIdx.x * blockDim.x + threadIdx.x;
	
	/*This will initialize the shared memory*/
	for(int i = thread; i < num_buckets; i += blockdim) {
		sharedHistogram[i].d_cnt = 0;
	}
	
	left = atom_list[block];
	__syncthreads();
	
	if(block < width) 
		for(int i = blockid + 1; i < num_blocks; i++) {		
			sharedAtomList[thread] = atom_list[i * blockdim + thread];
			__syncthreads();

			if(i*blockdim < width)
			for(int j = 0; j < blockdim; j++) {
				if(i*blockdim + j < width) {
					right = sharedAtomList[j];
					//double distance =sqrt(pow(left.x_pos - right.x_pos,2) + pow(left.y_pos - right.y_pos,2) + pow(left.z_pos - right.z_pos,2));
					double distance = sqrt((left.x_pos - right.x_pos)*(left.x_pos - right.x_pos) + (left.y_pos - right.y_pos)*(left.y_pos - right.y_pos) + (left.z_pos - right.z_pos)*(left.z_pos -right.z_pos));
					atomicAdd(&sharedHistogram[(int)(distance/size)].d_cnt,1);
				}
			}
			__syncthreads();
		}
	
	sharedAtomList[thread] = left;
	__syncthreads();

	if(block < width)
		for(int i = thread + 1; i < blockdim; i++) {
			
			if(blockdim * blockid + i < width) {
				right = sharedAtomList[i];
				//double distance =sqrt(pow(left.x_pos - right.x_pos,2) + pow(left.y_pos - right.y_pos,2) + pow(left.z_pos - right.z_pos,2));
				double distance = sqrt((left.x_pos - right.x_pos)*(left.x_pos - right.x_pos) + (left.y_pos - right.y_pos)*(left.y_pos - right.y_pos) + (left.z_pos - right.z_pos)*(left.z_pos - right.z_pos));
				atomicAdd(&sharedHistogram[(int)(distance/size)].d_cnt,1);
			}
		}
	__syncthreads();

	for(int i = thread; i < num_buckets; i += blockdim) {
		atomicAdd(&histogram[i].d_cnt,sharedHistogram[i].d_cnt);
	}
}


/* 
	set a checkpoint and show the (natural) running time in seconds 
*/
double report_running_time(int flag = 0) {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	if (flag == 0){
		printf("Running time for GPU Old version: %ld.%06ld\n", sec_diff, usec_diff);
	}
	else{
		printf("Running time for GPU new version: %ld.%06ld\n", sec_diff, usec_diff);
	}
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}


/* 
	print the counts in all buckets of the histogram 
*/
void output_histogram(bucket * histogram){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram[i].d_cnt);
		total_cnt += histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}



void cuda_eror_check(hipError_t errors, const char out[]){
	if (errors != hipSuccess)
    {
        printf("There is something wrong with cuda %s, %s, \n", out, hipGetErrorString(errors));
        exit(EXIT_FAILURE);
    }
}

void report_diff(bucket *histogram_CPU, bucket *histogram_GPU){
	//TODO
	printf("Histogram Difference:\n");
    for(int i = 0; i < num_buckets; i++) {
        if(i%5 == 0) /* we print 5 buckets in a row */
            printf("\n%02d: ", i);
        printf("%15lld ", (histogram_GPU[i].d_cnt - histogram_CPU[i].d_cnt));
        if(i != num_buckets - 1)
            printf("| ");
    }
    printf("\n\n");
}


int main(int argc, char **argv)
{
	int i;

	if (argc != 4 ) {
		printf("Error: Invalid Input, Please input all three parameters: {#of_samples} {bucket_width} {block_size}\n\n");
		exit(0);
	}

	PDH_acnt = atoi(argv[1]);
	PDH_res	 = atof(argv[2]);
	//printf("args are %d and %f\n", PDH_acnt, PDH_res);

	dim3 threads(atoi(argv[3]));
	dim3 grid(ceil((float)PDH_acnt/threads.x));

	num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;
	histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);

	atom_list = (atom *)malloc(sizeof(atom)*PDH_acnt);

	
	srand(1);
	/* generate data following a uniform distribution */
	for(i = 0;  i < PDH_acnt; i++) {
		atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
	}

	bucket *device_histogram = NULL;
	atom *device_atomList = NULL;

	bucket *device_histogram1 = NULL;
	atom *device_atomList1 = NULL;

	size_t histogram_size = sizeof(bucket) * num_buckets;
	size_t atom_size = sizeof(atom)*PDH_acnt;


	/*Data Copy for Old method*/
	hipError_t histogram_error =  hipMalloc((void**) &device_histogram, histogram_size);
	cuda_eror_check(histogram_error,"Error during hipMalloc for Histogram");
	hipError_t atom_list_error =  hipMalloc((void**) &device_atomList, atom_size);
	cuda_eror_check(atom_list_error,"Error during hipMalloc for atom list");

	hipError_t histogram_memcpy_error = hipMemcpy(device_histogram, histogram, histogram_size, hipMemcpyHostToDevice);
	cuda_eror_check(histogram_memcpy_error,"Error during hipMemcpy for Histogram");

	hipError_t atom_list_memcpy_error = hipMemcpy(device_atomList, atom_list, atom_size, hipMemcpyHostToDevice);
	cuda_eror_check(atom_list_memcpy_error,"Error during hipMemcpy for atom list");


	/*Data Copy for New method*/
	hipError_t histogram_error1 =  hipMalloc((void**) &device_histogram1, histogram_size);
	cuda_eror_check(histogram_error1,"Error during hipMalloc for Histogram");
	hipError_t atom_list_error1 =  hipMalloc((void**) &device_atomList1, atom_size);
	cuda_eror_check(atom_list_error1,"Error during hipMalloc for atom list");

	hipError_t histogram_memcpy_error1 = hipMemcpy(device_histogram1, histogram, histogram_size, hipMemcpyHostToDevice);
	cuda_eror_check(histogram_memcpy_error1,"Error during hipMemcpy for Histogram");

	hipError_t atom_list_memcpy_error1 = hipMemcpy(device_atomList1, atom_list, atom_size, hipMemcpyHostToDevice);
	cuda_eror_check(atom_list_memcpy_error1,"Error during hipMemcpy for atom list");
	

	printf("\n************************************** GPU Old **********************************************\n");

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	/* start counting time */
	gettimeofday(&startTime, &Idunno);
	/* call CPU single thread version to compute the histogram */
	PDH_baseline <<<grid, threads>>> (device_histogram, device_atomList, PDH_res, PDH_acnt);

	double temp_time_1 = report_running_time();
	

	hipError_t cuda_memcpy_error = hipMemcpy(histogram, device_histogram, histogram_size, hipMemcpyDeviceToHost);
	cuda_eror_check(cuda_memcpy_error, "Error during hipMemcpy for Histogram to Host");
	
	

	
	output_histogram(histogram);
	printf("\n\n\n************************************** GPU Algo 3**********************************************\n");

	gettimeofday(&startTime, &Idunno);

	/*Call GPU Code*/
	PDH2D_basline_algorithm3<<<grid,threads,num_buckets*sizeof(bucket)>>>(device_histogram1, device_atomList1, PDH_acnt, PDH_res, grid.x, num_buckets);

	double temp_time_2 = report_running_time(1);
	
	bucket *device_histogram_temp = (bucket *) malloc(sizeof(bucket) * num_buckets);
	// cudaMemcpy(device_histogram_temp, device_histogram, histogram_size, cudaMemcpyDeviceToHost);

	//TODO
	hipError_t last_error = hipGetLastError();
	cuda_eror_check(last_error, "Check for last error by hipGetLastError");
	/* check the total running time */ 
	
	

	hipError_t cuda_memcpy_error1 = hipMemcpy(device_histogram_temp, device_histogram, histogram_size, hipMemcpyDeviceToHost);
	cuda_eror_check(cuda_memcpy_error1, "Error during hipMemcpy for Histogram to Host");
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	
	
	/* print out the histogram */
	output_histogram(device_histogram_temp);

	printf("\n**********Time to generate:: %0.5f ms*********** \n\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	/*
	printf("\n\n********************************** Difference **************************************************\n");
	printf("The Difference between Old and New GPU Time : %lf\n", temp_time_2 - temp_time_1);
	report_diff(histogram, device_histogram_temp);
	*/

	cuda_eror_check(hipFree(device_histogram), "Free Device Historgram");
	cuda_eror_check(hipFree(device_atomList),"Free Device Atom List");

	cuda_eror_check(hipFree(device_histogram1), "Free Device Historgram");
	cuda_eror_check(hipFree(device_atomList1),"Free Device Atom List");
	
	hipFree(histogram);
	hipFree(atom_list);
	hipFree(device_histogram_temp);

	cuda_eror_check(hipDeviceReset(), "Reset the Device");

	return 0;
}


